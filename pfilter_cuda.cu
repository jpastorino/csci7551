#include "hip/hip_runtime.h"
//-----------------------------------------------------------------------
// Particle Filters Simulation   ---  SEQUENTIAL ALGORITHM
//-----------------------------------------------------------------------
//  Written by: Javier Pastorino
//  Updated in Dec-2016
//-----------------------------------------------------------------------
#include <iostream>
#include <iomanip>
#include <cmath>
#include <time.h>
#include <cstdlib>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;

//-----------------------------------------------------------------------
//   Data Structures and Constants
//-----------------------------------------------------------------------

#define THREADS_PER_BLOCK 1024

#define MAX_PART_TO_PRINT 20

struct Particle {
	int x,y;
	double weight;
	bool choosen;
};

struct Robot {
	int x,y;
};

Robot theRobot;  /*Variable that register the real position of the robot. Used to simulate the probabilities of the particles and compare if the algorithm succeeds.*/

bool showOutput=false;

/********************************************************************/
bool GetUserInput(int argc, char *argv[],int& spaceDimention, long& particleQuantity){
	/*Gets the parameters from input*/
	bool isOK = true;
	int robotX,robotY;

	if(argc < 4) 
	{
		cout << "Arguments:<spaceDimention> <particleQuantity> <showOutput>" << endl;
		cout << "spaceDimention : Space Matrix size [ spaceDimention X spaceDimention]" << endl;
		cout << "particleQuantity : number of particles to create" << endl;
		cout << "showOutput : 0|1 if show iteration output. Summary will always be shown." << endl;
		isOK = false;
	}
	else 
	{
		//get spaceDimention
		spaceDimention = atoi(argv[1]);
		if (spaceDimention <=0) 
		{
			cout << "Space size must be larger than 0" <<endl;
			isOK = false;
		}
		//get particleQuantity
		particleQuantity = atol(argv[2]);
		if (particleQuantity <= 0) 
		{
			cout << "Particles must be more than 0" <<endl;
			isOK = false;
		}

		//get showOutput
		int SO = atoi(argv[3]);
		if (SO ==0)	showOutput=false;
		else 
			if (SO ==1)	showOutput=true;
			else {
				cout << "showOutput should be 0 or 1" <<endl;
				isOK = false;
			}


		if (isOK){
			/*Read Robot Initial Location*/
			cout<<"Select robot position X (0.."<<spaceDimention-1<<"): ";  cin>>robotX;
			cout<<"Select robot position Y (0.."<<spaceDimention-1<<"): ";  cin>>robotY;
			
			theRobot.x=robotX;	theRobot.y=robotY;
		}
	}
	return isOK;
}


/********************************************************************/
void updateRobotMovements(int &dx, int &dy){
  	srand (time(NULL));/* initialize random seed: */

	if ( (rand()%100 ) > 50) {
		dx = 1;
		dy=0; 
	}   
	else {
		dx = 0;
		dy=1;
	}
}


/**************************** Run in CPU ****************************************/
double cpuEstimateParticleWeight(Particle aParticle){
	/* Estimates the weigh of a particle being in the robots position. */
	double distance = sqrt( pow( theRobot.x - aParticle.x ,2) + pow( theRobot.y - aParticle.y  ,2) );

	if (distance == 0)
		return 1;
	else
		return 1/distance;
}

/*****************************  Run in CPU ***************************************/
void estimateParticlesWeight(Particle* particleSpace, long  particleQuantity){
	/*For first initialization calculates the particle weight.  Could be improved in parallel*/

	double normWeight=0;

	for (long i=0;i<particleQuantity;i++){
		particleSpace[i].weight = cpuEstimateParticleWeight(particleSpace[i]);
		normWeight += particleSpace[i].weight;
	}

	
	for (long i=0; i<particleQuantity; i++){
		normWeight += particleSpace[i].weight;
	}

	for (long i=0; i<particleQuantity; i++){
		particleSpace[i].weight = (1 / normWeight) * particleSpace[i].weight;
	}

}


/********************************************************************/
void drawFirstParticleSet(Particle* particleSpace, int spaceDimention, long particleQuantity){
	/*Draw the first <particleQuantity> particles inside the matrix [spaceDimention X spaceDimention]*/
	int x,y;

  	srand (time(NULL));/* initialize random seed: */

	for ( int i=0; i<particleQuantity; i++){
 		x = rand() % spaceDimention;/* generate secret number between 1 and spaceDimention: */
 		y = rand() % spaceDimention;/* generate secret number between 1 and spaceDimention: */
 		particleSpace[i].x=x;
 		particleSpace[i].y=y;
 		particleSpace[i].weight=0;
 		particleSpace[i].choosen=false;
	}
}


/********************************************************************/
double calculateRandomProbability(long particleQuantity){
	double randomProbability=0;
	srand (time(NULL));/* initialize random seed: */
	randomProbability = (double)(((rand() << 15) + rand()) & ((1 << 24) - 1)) / (1 << 24);
	randomProbability = randomProbability * ( (double)1 / (double) particleQuantity );
	return randomProbability;
}

		

/********************************************************************/
void printMatrixParticles (Particle* particleSpace, long particleQuantity, int spaceDimention){
	
	/*Prints the current particles to screen*/
	if (spaceDimention <= MAX_PART_TO_PRINT && showOutput){
		long particlesUnderRobot=0;

		for (int i=0; i < spaceDimention; i++){
			printf("ROW [%2i] ", i );
			for (int j=0; j < spaceDimention; j++){
				long count=0;

				for ( int k = 0; k< particleQuantity; k++ ){
					if ( particleSpace[k].x == i && particleSpace[k].y == j)	
						count++;
				}

				if ( theRobot.x == i && theRobot.y == j){
					cout << "R.";
					particlesUnderRobot=count;
				}
				else{
					if (count >0) printf("%2i ", count); 
					else cout <<"--";
				}

			}
			cout << endl;
		}
		printf("Particles Under Robot: %2i ", particlesUnderRobot); 
	}
}


/********************************************************************/
void printParticles (Particle* particleSpace, long particleQuantity, int spaceDimention){
	/*Prints the current particles to screen*/
	if (spaceDimention <= MAX_PART_TO_PRINT && showOutput){
		for ( int i = 0; i < particleQuantity; i++ )
		    cout<<"particle "<< i << " X:"<<particleSpace[i].x << " Y:" << particleSpace[i].y << " Weight:"<< particleSpace[i].weight<<" Choosen:"<<particleSpace[i].choosen<<endl;
	}
}


/********************************************************************/
void displayInitialConfiguration(int spaceDimention, long particleQuantity){
	system ("clear");
	cout << "Simulation Configuration:" <<endl << "Space Dimention:" <<spaceDimention<<endl <<"Number of Particles:"<<particleQuantity<<endl;
	cout << "Robot initial position (x,y) = ("<<theRobot.x<<","<<theRobot.y<<")"<<endl; 
	cout <<"---------------------------------------------------"<<endl;
	cout <<"Press any key to start...\n";
	std::cin.ignore();
}


/********************************************************************/
void printParticleProbability(Particle* particleSpace, long particleQuantity , int spaceDimention){
	/* Prints a summary of the particles and the probabylity for each one*/

	long      summaryQty = 0;										//Stores the numeber of particles.
	double   *particleProbability = new double[particleQuantity]; 	//Stores the probability
	long     *particleNumber = new long[particleQuantity]; 			//Stores the number of particles in an specific cell
	Particle *particleSummary = new Particle[particleQuantity];  	//Stores the particles 

	for (long i=0; i < particleQuantity; i++){
		particleProbability[i]=0;
		particleNumber[i]=0;
		particleSummary[i].x = -1;
		particleSummary[i].y = -1;
	}


	for (long i=0; i<particleQuantity; i++){
		bool found=false;
		long index=0;
		long j=0;
		while (j<summaryQty and !found ){  /*Search for the particle in the Summary array*/
			if ( particleSpace[i].x == particleSummary[j].x 
				 && 
				 particleSpace[i].y == particleSummary[j].y    ){
				found=true;
				index=j;
			}
			j++;
		}

		if (! found){ /*must add it*/
			index=summaryQty;
			particleSummary[index].x = particleSpace[i].x;
			particleSummary[index].y = particleSpace[i].y;
			summaryQty++;
		}
		particleProbability[index] += particleSpace[i].weight;
		particleNumber[index] ++;
	}

	system ("clear");
	cout << "Simulation Configuration:" <<endl << "Space Dimention:" <<spaceDimention<<endl <<"Number of Particles:"<<particleQuantity<<endl;
	cout << "Robot Final position (x,y) = ("<<theRobot.x<<","<<theRobot.y<<")"<<endl; 
	cout <<"---------------------------------------------------"<<endl;

	cout<<"Particle Summary:\n";
	for (long j=0; j < summaryQty; j++){
		printf("Position (%3i) (x,y)=(%5i,%5i) #Particles: %7i Probability of robot here: %6f%% \n", j,particleSummary[j].x,particleSummary[j].y,particleNumber[j],((double)particleProbability[j]*100) );
	}

	delete[] particleProbability;
	delete[] particleNumber;
	delete[] particleSummary;
}


/******************************** RUNS IN GPU ************************************/
__global__ void normalizeWeights(Particle* particleSpace, long particleQuantity){  
	/*Normalizes the weight of the particles so Sum(Wi)==1.  we calculate the normalization factor as the sum of the weight. */
	unsigned int threadId = ( blockIdx.x * blockDim.x ) + threadIdx.x;

	__shared__ double normWeight;

	if (threadId == 0){
		//do SERIAL Reduction
		normWeight=0;
		for (long i=0; i<particleQuantity; i++)
			normWeight+=particleSpace[i].weight;
	}
	__syncthreads();

	if (threadId<particleQuantity)
		particleSpace[threadId].weight = (1 / normWeight) * particleSpace[threadId].weight;
	
}


/******************************** RUNS IN GPU ************************************/
__global__ void prefixCalculation(Particle *particles, double *prefix, long N){
	unsigned int threadId = ( blockIdx.x * blockDim.x ) + threadIdx.x;

	if (threadId == 0){  //Dummy in serial to check everything.
		//*Calculating the cumulative weight -- Prefix ****
		prefix[0]=particles[0].weight;
		for (long k=1; k<N; k++){	
			prefix[k] = prefix[k-1]+particles[k].weight;
		}
	} 
}




/**************************** Run in GPU ****************************************/
__device__ double estimateParticleWeight(Particle aParticle, Robot theRobot){
	/* Estimates the weigh of a particle being in the robots position. */
	double distance = sqrt( (double)(pow( (double)(theRobot.x - aParticle.x) ,2) + pow( (double)(theRobot.y - aParticle.y)  ,2)) );

	if (distance == 0)
		return 1;
	else
		return 1/distance;
}

/******************************** RUNS IN GPU ************************************/
__global__ void applyParticleFilters(Particle* particleSpace, double* cumulativeWeight, int spaceDimention, long particleQuantity, int dx, int dy, Robot theRobot, double randomProbability){

	unsigned int threadId = ( blockIdx.x * blockDim.x ) + threadIdx.x;
	long j=0;
	 
	if (threadId < particleQuantity)   //Control access in particle array boundaries
	{
 		double uk = (double) randomProbability +  ( (double) threadId / particleQuantity ) ;

		while (uk > cumulativeWeight[j] && j<particleQuantity-1) {	j ++;	}

		//APPLY RANDOM MOVEMENT.
		particleSpace[threadId].x = particleSpace[j].x + dx;
		particleSpace[threadId].y = particleSpace[j].y + dy;
		particleSpace[threadId].choosen = true;

		//Boundary control.
		if (particleSpace[threadId].x < 0 || 
			particleSpace[threadId].x > spaceDimention-1)	{
			particleSpace[threadId].x = particleSpace[threadId].x + (dx * - 2);	
		}
		if (particleSpace[threadId].y < 0 || 
			particleSpace[threadId].y > spaceDimention-1)	{
			particleSpace[threadId].y = particleSpace[threadId].y + (dy * - 2);	
		}

		particleSpace[threadId].weight = estimateParticleWeight(particleSpace[threadId], theRobot);
	}
}





//********************************************************************
// Main Program
//********************************************************************
int main(int argc, char *argv[])
{
	/*********************/
	/**** Variables      */
	int dx,dy;						//Robot movements
	int spaceDimention;
	long particleQuantity;
	Particle *particleSpace;
	Particle *d_particleSpace;
		double *cumulativeWeight;  

	double *d_cumulativeWeight;  
	double randomProbability;


	float runtime;

	if ( GetUserInput(argc,argv,spaceDimention,particleQuantity) == false ) return 1;  

	//Configure GPU Thread distribution.
	int numOfBlocks = particleQuantity / THREADS_PER_BLOCK + ((particleQuantity%THREADS_PER_BLOCK)?1:0);

	displayInitialConfiguration(spaceDimention,particleQuantity);  /*Prints initial configuration.*/
	cout <<"RUNNIN CUDA WITH BLOCK:"<<numOfBlocks<<"  AND THREADS:"<<THREADS_PER_BLOCK<<endl;

	runtime = clock()/(float)CLOCKS_PER_SEC;
	
	//Initialize the robot movements.
	updateRobotMovements(dx,dy);

	particleSpace = new Particle[particleQuantity];  /*Allocates memory for the particles.*/ 
	cumulativeWeight=new double[particleQuantity];
	
	//Allocate memory on device for the particles and prefix.
	hipMalloc((void**)&d_particleSpace,    particleQuantity*sizeof(Particle));
	hipMalloc((void**)&d_cumulativeWeight, particleQuantity*sizeof(double));  //Prefix Calculation.
 
	drawFirstParticleSet(particleSpace,spaceDimention,particleQuantity); /*draw the first set of particles sparsed on the grid.*/

	estimateParticlesWeight(particleSpace,particleQuantity);//<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<

	printMatrixParticles(particleSpace,particleQuantity,spaceDimention); /*prints the inital particles (depending on size)*/
	printParticles(particleSpace,particleQuantity,spaceDimention);       /*prints the inital particles (depending on size)*/


	//Copy Particles to the GPU. After that all procedure will occur on the GPU until finished.
	hipMemcpy(d_particleSpace, particleSpace, particleQuantity*sizeof(Particle), hipMemcpyHostToDevice);

	/*ITERATE!!!*/
	int iterationsToDo=0;

	while (iterationsToDo < (spaceDimention*0.5) ){

		if (iterationsToDo % (spaceDimention/2) == 0 )
			updateRobotMovements(dx,dy);

		if (showOutput){	cout <<"Iteration No."<<iterationsToDo<<" Press any key to continue...\n";	std::cin.ignore();	}

		theRobot.x = theRobot.x + dx;	theRobot.y = theRobot.y + dy;	/*Robot Moves.*/

		//Boundary control.
		if (theRobot.x < 0 || theRobot.x > spaceDimention-1)	{theRobot.x = theRobot.x + (dx * -2);	dx = dx * -1;}
		if (theRobot.y < 0 || theRobot.y > spaceDimention-1)	{theRobot.y = theRobot.y + (dy * -2);	dy = dy * -1;}
		
		prefixCalculation<<<numOfBlocks, THREADS_PER_BLOCK>>>(d_particleSpace, d_cumulativeWeight, particleQuantity); 	//Lets Calculate Prefix, 
		hipDeviceSynchronize();  //SyncThreads to continue with the prefix calculated.

		randomProbability = calculateRandomProbability(particleQuantity);
		applyParticleFilters<<<numOfBlocks,THREADS_PER_BLOCK>>> (d_particleSpace, d_cumulativeWeight, spaceDimention, particleQuantity,dx,dy, theRobot, randomProbability);		//Lets apply the particle filter
		hipDeviceSynchronize();  //Sync threads to continue next step
		
		normalizeWeights<<<numOfBlocks,THREADS_PER_BLOCK>>> (d_particleSpace, particleQuantity);//<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
		hipDeviceSynchronize();  //Sync threads to continue to next iteration.

		//Could not print as data is in GPU,otherwise had to copyback:	printMatrixParticles(particleSpace,particleQuantity,spaceDimention); /*prints the inital particles (depending on size)*/

		iterationsToDo++;
	}
	//Copy back the data from GPU to CPU.
	hipMemcpy(particleSpace, d_particleSpace, particleQuantity*sizeof(Particle), hipMemcpyDeviceToHost);

	
	//////// Display Information.
	cout<<endl;
	printMatrixParticles(particleSpace,particleQuantity,spaceDimention); 
	
	cout<<endl<<endl;
	printParticleProbability(particleSpace, particleQuantity, spaceDimention );

	cout<<endl<<endl;
	runtime = clock()/(float)CLOCKS_PER_SEC - runtime;
	cout<< "Program runs in " << setiosflags(ios::fixed) << setprecision(2) << runtime << " seconds\n"; 

	hipFree(d_particleSpace);
	hipFree(d_cumulativeWeight);
	delete[] particleSpace;

	cout <<"---------------------------------------------------"<<endl;
	cout <<"-----            Simulation Ended              ----"<<endl;
	cout <<"---------------------------------------------------"<<endl;
	return 0;
}